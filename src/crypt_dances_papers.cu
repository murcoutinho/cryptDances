#include "hip/hip_runtime.h"
#include "arx_cryptanalysis.cuh"
#include "papers.cuh"
#include "util.cuh"

int my_rank, num_procs;

void forro_4round_linear_approximation(FILE *output_file)
{
    uint64_t N = 1;
    linear_approximation_t la = {0};

    N<<=38;
    la.correlation.expected = 0.0476;
    strcpy(la.correlation.paper,"[Coutinho 2022]");
    la.alg_type = ALG_TYPE_FORRO;
    la.correlation.number_of_trials = N;
    lob_define_single_bit(&la.input, 10,0,8);
    
    expand_linear_equation(&la, 8);

    compute_differential_or_linear_correlation(&la, TYPE_LINEAR);
    if(la.correlation.is_significant & my_rank==0)
      la_print(output_file, la);
}

void forro_5round_linear_approximation(FILE *output_file)
{
    uint64_t N = 1;
    linear_approximation_t L[15] = {0};
    double expected[5] = {0.0278, 0.1667, 0.0046, 0.0046, 0.000284};

    N<<=38;
    L[0].input.subround = 8;
    set_bit(L[0].input.mask, 10, 0);
    
    for(int i=0;i<13;i++)
    {
        L[i].alg_type = ALG_TYPE_FORRO;
        strcpy(L[i].correlation.paper,"[Coutinho 2022]");
        expand_linear_equation(&L[i],1);
        if(i>=8)
        {
            L[i].correlation.number_of_trials = N;
            L[i].correlation.expected = expected[i-8];
            compute_differential_or_linear_correlation(&L[i], TYPE_LINEAR);
            if(L[i].correlation.is_significant & my_rank == 0)
                la_print(output_file, L[i]);
        }
        memcpy(L[i+1].input.mask, L[i].output.mask, sizeof(uint32_t)*MAXIMUM_STATE_SIZE);
        L[i+1].input.subround = L[i].output.subround;
    }
}


void salsa_pnb_attack_using_ble(FILE *output_file)
{
    int differential_E1_part_subrounds = 1, differential_E2_part_subrounds = 3;
    int k = 0, alg_type = ALG_TYPE_SALSA;
    int idw = 7;
    int idb = 31;
    int odw = 4;
    int odb = 7; 
    pnb_t pnb = {0};
    difference_t od = {{0}, {0,4,12},{0,7,0},4, 3};
    lob_compute_mask_from_list_of_bits(&od);
    algorithm alg;
    define_alg(&alg, alg_type);

    pnb.subrounds = 7;
    pnb.threshold = 0.3;
    pnb.statistic_type = STAT_MEDIAN;
    pnb.correlation_of_g.number_of_trials = 1;
    pnb.correlation_of_g.number_of_trials <<= 34;
    pnb_define_alg(&pnb, alg_type);
    
    //Use Lipmaa and Moriai to compute first round differential
    lob_define_single_bit(&pnb.diff.input, idw, idb, 0);
    alg.differential_update(pnb.diff.input.mask, differential_E1_part_subrounds, &k);
    lob_compute_list_of_bits_from_mask(&pnb.diff.input);
    pnb.diff.input.subround = differential_E1_part_subrounds;

    //Compute differential correlation for each bit from the backward expansion
    double differential_correlation = 1;
    for(int i=0;i<od.number_of_bits;i++)
    {
        memset(&pnb.diff.output, 0, sizeof(difference_t));
        lob_define_single_bit(&pnb.diff.output, od.words[i], od.bits[i], 
            differential_E2_part_subrounds + differential_E1_part_subrounds);
        search_until_find_correlation(&pnb.diff, TYPE_DIFFERENTIAL);
        differential_correlation *= pnb.diff.correlation.observed;
    }
    pnb.diff.correlation.observed = differential_correlation;
    memcpy(&pnb.diff.output, &od, sizeof(difference_t));

    //Set linear expansion with correlation 1
    memcpy(&pnb.la.input, &pnb.diff.output, sizeof(diff_lin_t));
    pnb.la.correlation.observed = 1;
    lob_define_single_bit(&pnb.la.output, odw, odb,pnb.la.input.subround+1);

    //Attack 7 and 8 rounds
    for(int sr=7; sr<=8; sr++)
    {
        pnb.subrounds = sr;
        compute_neutrality_vector(&pnb, (1<<26));
        get_pnb_list(&pnb);
        compute_correlation_of_g(&pnb);
        compute_complexity_of_the_attack(&pnb);
        pnb.data_complexity += k;
        pnb.time_complexity += k;
        if(my_rank == 0)
            pnb_print(output_file, pnb);
    }
}

void euro2022(FILE *output_file)
{
    pnb_t pnb = {0};
    algorithm alg;
    int diff_E1_subrounds = 2, diff_E2_subrounds = 5, linear_part_subrounds = 1;
    int k=0, alg_type = ALG_TYPE_CHACHA;

    int idw = 13, idb = 6, odw = 2, odb = 0;
    pnb.subrounds = 14;
    pnb.threshold = 0.2;
    pnb.alg_type = alg_type;
    pnb.statistic_type = STAT_MEDIAN;
    uint64_t number_of_trials_for_neutrality = (1<<28), number_of_trials_for_bias_of_g = 1;
    number_of_trials_for_bias_of_g<<=31;

    //Use Lipmaa and Moriai to compute first round differential
    define_alg(&alg, alg_type);
    lob_define_single_bit(&pnb.diff.input, idw, idb, 0);
    alg.differential_update(pnb.diff.input.mask, diff_E1_subrounds, &k);
    lob_compute_list_of_bits_from_mask(&pnb.diff.input);
    pnb.diff.input.subround = diff_E1_subrounds;

    lob_define_single_bit(&pnb.diff.output, odw, odb, diff_E2_subrounds + diff_E1_subrounds);
    search_until_find_correlation(&pnb.diff, TYPE_DIFFERENTIAL);
    la_compute_from_differential(&pnb.la, pnb.diff, linear_part_subrounds);

    compute_neutrality_vector(&pnb, number_of_trials_for_neutrality);
    get_pnb_list(&pnb);

    pnb_remove(&pnb, 49); pnb_remove(&pnb, 51); pnb_remove(&pnb, 52);
    pnb_remove(&pnb, 59); pnb_remove(&pnb, 169);

    pnb_iteractive_selection(&pnb, 0.45, number_of_trials_for_bias_of_g, 12);
    pnb.correlation_of_g.number_of_trials = number_of_trials_for_bias_of_g;

    compute_correlation_of_g(&pnb);
    compute_complexity_of_the_attack(&pnb);

    if(my_rank == 0)
        pnb_print(output_file, pnb);
}


void differential_results(FILE *output_file)
{
    int count = 0;
    while(1)
    {
        differential_t diff = paperdiff[count];

        if(strcmp(diff.correlation.paper, "Stop")==0)
            break;

        lob_compute_mask_from_list_of_bits(&(diff.input));
        lob_compute_mask_from_list_of_bits(&(diff.output));
        search_until_find_correlation(&diff, TYPE_DIFFERENTIAL);
        if(my_rank == 0)
            differential_print(output_file, diff);

        count++;
    }
}

void linear_results(FILE *output_file)
{
    int count = 0;
    while(1)
    {
        linear_approximation_t lin_approx = paperlin[count];

        if(strcmp(lin_approx.correlation.paper, "Stop")==0)
            break;

        lob_compute_mask_from_list_of_bits(&(lin_approx.input));
        lob_compute_mask_from_list_of_bits(&(lin_approx.output));
        search_until_find_correlation(&lin_approx, TYPE_LINEAR);
        if(my_rank == 0)
            la_print(output_file, lin_approx);

        count++;
    }

#ifdef COUTINHO_2022_FORRO_LINEAR_APPROXIMATIONS
    forro_4round_linear_approximation(output_file);
    forro_5round_linear_approximation(output_file);
#endif
}

void pnb_results(FILE *output_file)
{
    uint64_t number_of_trials_for_neutrality = (1<<30), number_of_trials_for_neutrality_for_bias_of_g = 1;
    number_of_trials_for_neutrality_for_bias_of_g <<= 34;

#ifdef AUMASSON_2008_PNB
    //TABLE 2
    pnb_attack_for_single_bit_differential(7,31,1,14,7,4,0,0.9, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_SALSA, STAT_MEDIAN,output_file);
    pnb_attack_for_single_bit_differential(7,31,1,14,7,4,0,0.8, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_SALSA, STAT_MEDIAN, output_file);
    pnb_attack_for_single_bit_differential(7,31,1,14,7,4,0,0.7, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_SALSA, STAT_MEDIAN, output_file);
    pnb_attack_for_single_bit_differential(7,31,1,14,7,4,0,0.6, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_SALSA, STAT_MEDIAN, output_file);
    pnb_attack_for_single_bit_differential(7,31,1,14,7,4,0,0.5, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_SALSA, STAT_MEDIAN, output_file);

    //Attacks sec 3.5
    pnb_attack_for_single_bit_differential(7,31,1,14,8,4,0,0.12, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_SALSA, STAT_MEDIAN, output_file);
    pnb_attack_for_single_bit_differential(13,13,11,0,12,6,0,0.6, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_CHACHA, STAT_MEDIAN, output_file);
    pnb_attack_for_single_bit_differential(13,13,11,0,14,6,0,0.5, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_CHACHA, STAT_MEDIAN, output_file);
#endif

#ifdef CHOUDHURI_2016_PNB
    pnb_attack_for_single_bit_differential(12,21,2,0,12,6,2,0.4, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_CHACHA, STAT_MEDIAN, output_file);
    pnb_attack_for_single_bit_differential(12,21,2,0,12,6,0,0.4, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_CHACHA, STAT_MEDIAN, output_file);
#endif

#ifdef COUTINHO_2020_PNB
    pnb_attack_for_single_bit_differential(14,6,3,0,12,7,1,0.4, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_CHACHA, STAT_MEDIAN, output_file);
    pnb_attack_for_single_bit_differential(14,6,3,0,14,7,1,0.35, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_CHACHA, STAT_MEDIAN, output_file);
#endif
#ifdef COUTINHO_2022_FORRO_PNB
    pnb_attack_for_single_bit_differential(5,11,10,0,20,8,0,0.25, number_of_trials_for_neutrality,
        number_of_trials_for_neutrality_for_bias_of_g, ALG_TYPE_FORRO, STAT_MEDIAN, output_file);
#endif
#ifdef COUTINHO_2022_SALSA_PNB
    salsa_pnb_attack_using_ble(output_file);
#endif
#ifdef DEY_2022_CHACHA_PNB
    euro2022(output_file);
#endif
}

int main()
{
    FILE *p = NULL;
    
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &my_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    if(my_rank == 0) {
        create_folder_if_doesnt_exist("results/");
        p = fopen("results/paper_results.dat", "w");
    }
    differential_results(p);
    linear_results(p);
    pnb_results(p);

    if(my_rank == 0)
        fclose(p);

    MPI_Finalize();
    return 0;
}
